
#include <hip/hip_runtime.h>
#define TILE_DIM 32

__global__ void matmul_kernel(float* pfMatrixA, float* pfMatrixB, float* pfMatrixC, int m, int k, int n)
{
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for(int i =0; i < k; i++)
    {
        sum += pfMatrixA[nRow * k + i] * pfMatrixB[i * n + nCol];
    }
    pfMatrixC[nRow * n + nCol] = sum;
}

void launch_matmul(
                    float* array_A,
                    float* array_B,
                    float* array_C,
                    int M,
                    int K,
                    int N
                    )
{
    dim3 block_size(TILE_DIM, TILE_DIM);
    dim3 grid_size((M + TILE_DIM - 1) / TILE_DIM, (N + TILE_DIM - 1) / TILE_DIM);
    matmul_kernel<<<grid_size, block_size>>>(array_A, array_B, array_C, M, K, N);
}
